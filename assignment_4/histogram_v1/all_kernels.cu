#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>
#include "timer.h"
#include <unistd.h>
#include <getopt.h>
using namespace std;


// same as sequential, single thread, only overhead
__global__ void histogramKernel(unsigned char* image, long img_size, unsigned int* histogram, int hist_size) {
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid == 1){
        for (int i=0; i<img_size; i++) {
            histogram[image[i]]++;
        }
    }

}