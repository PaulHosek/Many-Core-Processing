#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <float.h>

extern "C"
{
    #include "input.h"
    #include "output.h"
}
#include "cuda_compute.h"

#define BLOCK_SIZE_X 32 
#define BLOCK_SIZE_Y 32 
#define BLOCK_SIZE 1024 
#define WARP_SIZE 32

#define C_CDIR (0.25 * sqrt(2.0) / (sqrt(2.0) + 1.0))
#define C_CDIAG (0.25 / (sqrt(2.0) + 1.0))

__global__ void update_temperatures(double * temperature_old, double * temperature_new, double * conductivity, const int M, const int N)
{
    double old_temperature, new_temperature, c_c, rest_c_c;
    int inside_grid = 0;
    const int tidx = threadIdx.x;
    const int tidy = threadIdx.y;

    __shared__ double temperatures[BLOCK_SIZE_Y][BLOCK_SIZE_X][2];

    const int row = blockIdx.y * (blockDim.y - 2) + threadIdx.y;
    int col = blockIdx.x * (blockDim.x -2) + threadIdx.x - 1;
    inside_grid = (col >= 0) && (col < M) && (row < N + 1) && row;
    col += (col < 0) * M - (col == M) * M;
    const int index = row * M + col;

    if ((row < N + 2) && (col < M)) temperatures[tidy][tidx] = temperature_old[index];
    __syncthreads();

    if ((tidx) && (tidx < BLOCK_SIZE_X-1) && (tidy) && (tidy < BLOCK_SIZE_Y-1) && inside_grid)
    {
        //if (((tidx == BLOCK_SIZE_X-2) && (tidy == BLOCK_SIZE_Y-2)) || ((row == N) && (col == M-1))) printf("row=%d, col=%d, index=%d\n",row,col,index);
        old_temperature = temperature_old[index];
        c_c = conductivity[index - M];
        rest_c_c = 1 - c_c;

        new_temperature = old_temperature * c_c;
        new_temperature += (temperatures[tidy][tidx-1] + temperatures[tidy][tidx+1] +
                temperatures[tidy-1][tidx] + temperatures[tidy+1][tidx]) * rest_c_c * C_CDIR;
        new_temperature += (temperatures[tidy-1][tidx-1] + temperatures[tidy+1][tidx-1] +
                temperatures[tidy-1][tidx+1] + temperatures[tidy+1][tidx+1]) * rest_c_c * C_CDIAG;
        temperature_new[index] = new_temperature;
    }
}

__global__ void get_maxdiff_per_thread(double * temperature_old, double * temperature_new, double * maxdiff, const int M, const int N)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < M*N) 
    {
        maxdiff[index] = abs(temperature_old[index+M] - temperature_new[index+M]);
    }
}

__global__ void get_maxdiff_per_block(double * maxdiff, int offset, int grid_size)
{
    int index = (blockIdx.x * blockDim.x + threadIdx.x) * offset;
    const int in_grid = index < grid_size;

    double local_maxdiff, diff;

    __shared__ double s_max_diff[BLOCK_SIZE];

    s_max_diff[threadIdx.x] = maxdiff[in_grid * index];

    __syncthreads();

    for (unsigned int s = BLOCK_SIZE/2; s>=WARP_SIZE; s/=2) 
    {
        if (threadIdx.x < s)
        { 
            local_maxdiff = s_max_diff[threadIdx.x];

            if ((diff = s_max_diff[threadIdx.x + s]) > local_maxdiff) 
            {
                s_max_diff[threadIdx.x] = diff;
            }
        }
        __syncthreads();
    }

    if (threadIdx.x < WARP_SIZE)
    {
        local_maxdiff = s_max_diff[threadIdx.x];
        for (unsigned int s = WARP_SIZE/2; s > 0; s /= 2)
        {
            if ((diff = __shfl_down_sync(0xffffffff, local_maxdiff, s)) > local_maxdiff) 
            {
                local_maxdiff = diff;
            }
        }
    }

    if (!threadIdx.x)
    {
        maxdiff[index] = local_maxdiff;
    } 
}

__global__ void initialize_statistics(double * temperatures, double * min, double * max, double * sum, const int M, const int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    double temperature;

    if (index < M*N) 
    {
        temperature = temperatures[index + M];
        min[index] = temperature;
        max[index] = temperature;
        sum[index] = temperature;
    }
}

__global__ void get_statistics_per_block(double * min, double * max, double * sum, int offset, int grid_size)
{
    int index = (blockIdx.x * blockDim.x + threadIdx.x) * offset;
    double local_sum, local_min, local_max, new_min, new_max;
    const int in_grid = index < grid_size;

    __shared__ double s_sum[BLOCK_SIZE];
    __shared__ double s_min[BLOCK_SIZE];
    __shared__ double s_max[BLOCK_SIZE];

    s_min[threadIdx.x] = min[in_grid * index];
    s_max[threadIdx.x] = max[in_grid * index];
    s_sum[threadIdx.x] = in_grid * sum[in_grid * index];

    __syncthreads();

    for (unsigned int s = BLOCK_SIZE/2; s>=WARP_SIZE; s/=2) 
    {
        if (threadIdx.x < s)
        {
            local_min = s_min[threadIdx.x];
            local_max = s_max[threadIdx.x];

            if ((new_min = s_min[threadIdx.x + s]) < local_min) 
            {
                s_min[threadIdx.x] = new_min;
            }
            if ((new_max = s_max[threadIdx.x + s]) > local_max) 
            {
                s_max[threadIdx.x] = new_max;
            }
            s_sum[threadIdx.x] += s_sum[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x < WARP_SIZE)
    {
        local_sum = s_sum[threadIdx.x];
        local_min = s_min[threadIdx.x];
        local_max = s_max[threadIdx.x];
        for (unsigned int s = WARP_SIZE/2; s > 0; s /= 2)
        {
            if ((new_min = __shfl_down_sync(0xffffffff, local_min, s)) < local_min) 
            {
                local_min = new_min;
            }
            if ((new_max = __shfl_down_sync(0xffffffff, local_max, s)) > local_max) 
            {
                local_max = new_max;
            }
            local_sum += __shfl_down_sync(0xffffffff, local_sum, s);
        }
    }

    if (!threadIdx.x)
    {
        sum[index] = local_sum;
        min[index] = local_min;
        max[index] = local_max;
    } 
}

extern "C"
__host__ void print_grid(double const * grid, int m, int n)
{
    for(int i=0; i < n; i++)
    {
        for(int j=0; j < m; j++)
        {
            printf("%.2f ", grid[i*m+j]);
        }
        printf("\n\n");
    }
}

extern "C"
__host__ void initialize(const struct parameters* p, double* temperature_old, double* temperature_new, double* conductivity)
{
    int MN = p->N * p-> M; 
    int M = p->M;
    int index;
    // Halo rows
    for (index = 0; index < M; index++)
    {
        temperature_old[index] = p->tinit[index];
        temperature_new[index] = p->tinit[index];
        temperature_old[MN + M + index] = p->tinit[MN - M + index];
        temperature_new[MN + M + index] = p->tinit[MN - M + index];
    }
    // Fill the temperature values into the grid cells
    for (index = 0; index < MN; index++)
    {
        temperature_old[M + index] = p->tinit[index];
        conductivity[index] = p->conductivity[index];
    }
}

extern "C"
__host__ void cuda_do_compute(const struct parameters* p, struct results *r)
{
    // Initialize grid 
    const int N = p->N;
    const int M = p->M;
    double * temperature_old = (double *) malloc((N + 2) * M * sizeof(double));
    double * temperature_new = (double *) malloc((N + 2) * M * sizeof(double));
    double * conductivity = (double *) malloc(N * M * sizeof(double));
    double * d_temperature_old; 
    double * d_temperature_new; 
    double * d_conductivity;
    double * d_maxdiff;
    double * d_sum; 
    double * d_min; 
    double * d_max;
    int it = 1;
    const int grid_size = (p->N * p->M);
    int offset;
    const int maxiter = p->maxiter;
    const int period = p->period;
    const double threshold = p->threshold;
    int converged, compute_statistics, iters_to_next_period = period - 1;
    double maxdiff, tmin, tmax, tsum;
    hipError_t err;

    initialize(p, temperature_old, temperature_new, conductivity);

    //allocate GPU memory
    err = hipMalloc((void **)&d_temperature_old, (N + 2) * M * sizeof(double));
    if (err != hipSuccess) fprintf(stderr, "Error in hipMalloc d_temperature_old: %s\n", hipGetErrorString(err));
    err = hipMalloc((void **)&d_temperature_new, (N + 2) * M *sizeof(double));
    if (err != hipSuccess) fprintf(stderr, "Error in hipMalloc d_temperature_new: %s\n", hipGetErrorString(err));
    err = hipMalloc((void **)&d_conductivity, M*N*sizeof(double));
    if (err != hipSuccess) fprintf(stderr, "Error in hipMalloc d_conductivity: %s\n", hipGetErrorString(err));
    err = hipMalloc((void **)&d_maxdiff, M*N*sizeof(double));
    if (err != hipSuccess) fprintf(stderr, "Error in hipMalloc d_maxdiff: %s\n", hipGetErrorString(err));
    err = hipMalloc((void **)&d_min,  M*N*sizeof(double));
    if (err != hipSuccess) fprintf(stderr, "Error in hipMalloc d_min: %s\n", hipGetErrorString(err));
    err = hipMalloc((void **)&d_max,  M*N*sizeof(double));
    if (err != hipSuccess) fprintf(stderr, "Error in hipMalloc d_max: %s\n", hipGetErrorString(err));
    err = hipMalloc((void **)&d_sum,  M*N*sizeof(double));
    if (err != hipSuccess) fprintf(stderr, "Error in hipMalloc d_sum: %s\n", hipGetErrorString(err));
    
    // copy to GPU memory
    err = hipMemcpy(d_temperature_new, temperature_new, (N + 2) * M * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) fprintf(stderr, "Error in hipMemcpy d_temperature_new: %s\n", hipGetErrorString( err ));
    err = hipMemcpy(d_temperature_old, temperature_old, (N + 2) * M * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) fprintf(stderr, "Error in hipMemcpy d_temperature_old: %s\n", hipGetErrorString( err ));
    err = hipMemcpy(d_conductivity, conductivity, N * M * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) fprintf(stderr, "Error in hipMemcpy d_conductivity: %s\n", hipGetErrorString( err ));

    //setup the grid and thread blocks
    int nblocks_x = int(ceilf(M/(double)(BLOCK_SIZE_X-2)));//n divided by thread block size rounded up
    int nblocks_y = int(ceilf(N/(double)(BLOCK_SIZE_Y-2)));
    int nblocks_x_statistics = int(ceilf(grid_size / (double)BLOCK_SIZE));
    dim3 threads_temperatures(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
    dim3 threads_statistics(BLOCK_SIZE,1,1);
    dim3 grid_temperatures(nblocks_x, nblocks_y, 1);
    dim3 grid_statistics(nblocks_x_statistics,1,1);

    struct timespec before, after;
    hipDeviceSynchronize();
    clock_gettime(CLOCK_MONOTONIC, &before);

    do {
        // Check convergence every timestep
        converged = 0;
        compute_statistics = (!iters_to_next_period || it == maxiter);
        update_temperatures<<<grid_temperatures, threads_temperatures>>>(d_temperature_old, d_temperature_new, d_conductivity, M, N);
        hipDeviceSynchronize();
        //check to see if all went well
        err = hipGetLastError();
        if (err != hipSuccess) fprintf(stderr, "Error during kernel execution: %s\n", hipGetErrorString(err));
        grid_statistics.x = nblocks_x_statistics;
        get_maxdiff_per_thread<<<grid_statistics, threads_statistics>>>(d_temperature_old, d_temperature_new, d_maxdiff, M, N);
        hipDeviceSynchronize();
        err = hipGetLastError();
        if (err != hipSuccess) fprintf(stderr, "Error during kernel execution: %s\n", hipGetErrorString(err));
        offset=1;
        while (offset * BLOCK_SIZE < grid_size)
        {
            get_maxdiff_per_block<<<grid_statistics, threads_statistics>>>(d_maxdiff, offset, grid_size);
            hipDeviceSynchronize();
            err = hipGetLastError();
            if (err != hipSuccess) fprintf(stderr, "Error during kernel execution: %s\n", hipGetErrorString(err));
            offset *= BLOCK_SIZE;
            grid_statistics.x = int(ceilf(grid_statistics.x / (double)offset));
        };
        grid_statistics.x = 1; 
        get_maxdiff_per_block<<<grid_statistics, threads_statistics>>>(d_maxdiff, offset, grid_size);
        hipDeviceSynchronize();
        err = hipGetLastError();
        if (err != hipSuccess) fprintf(stderr, "Error during kernel execution: %s\n", hipGetErrorString(err));

        err = hipMemcpy(&maxdiff, d_maxdiff, sizeof(double), hipMemcpyDeviceToHost);
        if (err != hipSuccess) fprintf(stderr, "Error in hipMemcpy &maxdiff: %s\n", hipGetErrorString( err ));
        if (maxdiff <= threshold)
        {
            converged = 1;
        }

        if (converged || compute_statistics)
        {
            grid_statistics.x = nblocks_x_statistics;
            initialize_statistics<<<grid_statistics, threads_statistics>>>(d_temperature_new, d_min, d_max, d_sum, M, N);
            hipDeviceSynchronize();
            err = hipGetLastError();
            if (err != hipSuccess) fprintf(stderr, "Error during kernel execution: %s\n", hipGetErrorString(err));
            offset=1;
            while (offset * BLOCK_SIZE < grid_size)
            {
                grid_statistics.x = int(ceilf(grid_statistics.x / (double)offset));
                get_statistics_per_block<<<grid_statistics, threads_statistics>>>(d_min, d_max, d_sum, offset, grid_size);
                hipDeviceSynchronize();
                err = hipGetLastError();
                if (err != hipSuccess) fprintf(stderr, "Error during kernel execution: %s\n", hipGetErrorString(err));
                offset *= BLOCK_SIZE;
            };
            grid_statistics.x = 1; 
            get_statistics_per_block<<<grid_statistics, threads_statistics>>>(d_min, d_max, d_sum, offset, grid_size);
            hipDeviceSynchronize();
            err = hipGetLastError();
            if (err != hipSuccess) fprintf(stderr, "Error during kernel execution: %s\n", hipGetErrorString(err));

            err = hipMemcpy(&tmin, d_min, sizeof(double), hipMemcpyDeviceToHost);
            if (err != hipSuccess) fprintf(stderr, "Error in hipMemcpy &tmin: %s\n", hipGetErrorString( err ));
            err = hipMemcpy(&tmax, d_max, sizeof(double), hipMemcpyDeviceToHost);
            if (err != hipSuccess) fprintf(stderr, "Error in hipMemcpy &tmax: %s\n", hipGetErrorString( err ));
            err = hipMemcpy(&tsum, d_sum, sizeof(double), hipMemcpyDeviceToHost);
            if (err != hipSuccess) fprintf(stderr, "Error in hipMemcpy &tsum: %s\n", hipGetErrorString( err ));


            r->niter = it;
            r->tmin = tmin;
            r->tmax = tmax;
            r->tavg = tsum/grid_size;
            r->maxdiff = maxdiff;
            clock_gettime(CLOCK_MONOTONIC, &after);
            r->time = (double)(after.tv_sec - before.tv_sec) +
              (double)(after.tv_nsec - before.tv_nsec) / 1e9;
            
            if (it < maxiter && !converged && p->printreports){
                // Only call print if it's not the last iteration and the print-parameter is set 
                report_results(p,r);
            }
        }

        // Flip old and new values
        {double * tmp = d_temperature_old; d_temperature_old = d_temperature_new; d_temperature_new = tmp;} 
        it++;
        if (iters_to_next_period)
        {
            iters_to_next_period--;
        }
        else
        {
            iters_to_next_period += period-1;
        }
    } while ((it <= maxiter) && (!converged));
    
    clock_gettime(CLOCK_MONOTONIC, &after);
    r->time = (double)(after.tv_sec - before.tv_sec) +
              (double)(after.tv_nsec - before.tv_nsec) / 1e9;

    double flops_per_it = 12.0;
    double Flops = (double)p->N * (double)p->M * 
                    (double)(r->niter * flops_per_it +
                    (double)r->niter / p->period) / r->time;
                    
    //clean up GPU memory allocations
    hipFree(d_temperature_old);
    hipFree(d_temperature_new);
    hipFree(d_conductivity);
}

// static void checkCudaCall(hipError_t result) {
//     if (result != hipSuccess) {
//         printf("cuda error \n");
//         exit(1);
//     }
// }


// __global__ void vectorAddKernel(float* deviceA, float* deviceB, float* deviceResult) {
//     unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
// // insert operation here
//     deviceResult[i] = deviceA[i]+deviceB[i];
// }

// extern "C" 
// void cuda_do_compute() {
//     int threadBlockSize = 512;
//     int n=1024; 
//     float a[1024], b[1024], result[1024];
//     // allocate the vectors on the GPU
//     float* deviceA = NULL;
//     checkCudaCall(hipMalloc((void **) &deviceA, n * sizeof(float)));
//     if (deviceA == NULL) {
//         printf("Error in hipMalloc! \n");
// 	return;
//     }
//     float* deviceB = NULL;
//     checkCudaCall(hipMalloc((void **) &deviceB, n * sizeof(float)));
//     if (deviceB == NULL) {
//         checkCudaCall(hipFree(deviceA));
//         printf("Error in hipMalloc! \n");
//         return;
//     }
//     float* deviceResult = NULL;
//     checkCudaCall(hipMalloc((void **) &deviceResult, n * sizeof(float)));
//     if (deviceResult == NULL) {
//         checkCudaCall(hipFree(deviceA));
//         checkCudaCall(hipFree(deviceB));
//         printf("Error in hipMalloc! \n");
//         return;
//     }


//     // copy the original vectors to the GPU
//     checkCudaCall(hipMemcpy(deviceA, a, n*sizeof(float), hipMemcpyHostToDevice));
//     checkCudaCall(hipMemcpy(deviceB, b, n*sizeof(float), hipMemcpyHostToDevice));

//     // execute kernel
//     vectorAddKernel<<<n/threadBlockSize, threadBlockSize>>>(deviceA, deviceB, deviceResult);
//     hipDeviceSynchronize();

//     // check whether the kernel invocation was successful
//     checkCudaCall(hipGetLastError());

//     // copy result back
//     checkCudaCall(hipMemcpy(result, deviceResult, n * sizeof(float), hipMemcpyDeviceToHost));
//     checkCudaCall(hipMemcpy(b, deviceB, n * sizeof(float), hipMemcpyDeviceToHost));

//     checkCudaCall(hipFree(deviceA));
//     checkCudaCall(hipFree(deviceB));
//     checkCudaCall(hipFree(deviceResult));

// }
