#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>
#include "timer.h"
#include <unistd.h>
#include <getopt.h>

using namespace std;

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
void die(const char *msg){
    if (errno != 0) 
        perror(msg);
    else
        fprintf(stderr, "error: %s\n", msg);
    exit(1);
}   

void generate_image(int num_rows, int num_cols, unsigned char * image){
    for (int i = 0; i < num_cols * num_rows; ++i)
    {
        image[i] = (unsigned char) (rand() % 256); //255 + 1 for num bins
    }
}

void read_image(const char * image_path, int num_rows, int num_cols, unsigned char * image){
	char format[3];
    FILE *f;
    unsigned imgw, imgh, maxv, v;
    size_t i;

	printf("Reading PGM data from %s...\n", image_path);

	if (!(f = fopen(image_path, "r"))) die("fopen");

	fscanf(f, "%2s", format);
    if (format[0] != 'P' || format[1] != '2') die("only ASCII PGM input is supported");
    
    if (fscanf(f, "%u", &imgw) != 1 ||
        fscanf(f, "%u", &imgh) != 1 ||
        fscanf(f, "%u", &maxv) != 1) die("invalid input");

    if (imgw != num_cols || imgh != num_rows) {
        fprintf(stderr, "input data size (%ux%u) does not match cylinder size (%zux%zu)\n",
                imgw, imgh, num_cols, num_rows);
        die("invalid input");
    }

    for (i = 0; i < num_cols * num_rows; ++i)
    {
        if (fscanf(f, "%u", &v) != 1) die("invalid data");
        image[i] = (unsigned char) (((int)v * 255) / maxv); //255 for num bins
    }
    fclose(f);
}

static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}



__global__ void histogramKernel(unsigned char* image, long img_size, unsigned int* histogram, int hist_size) {
    // shared memory to store local histogram of warp
    __shared__ unsigned int smem_hist[256];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int lane = threadIdx.x & (32-1); // tid in warp
//    int warp_id = threadIdx.x >> 5; // warp index, = /32+ round down
    unsigned int local_hist[256] = {0};


    // process 32 elements
    // each thread processes every max(tid)s pixel. if there are 12 threads, every 12th pixel will be processed
    // Note, that if values in image have pattern that matches this division, collisions are maximal
    for (int i = tid; i < img_size; i += blockDim.x * gridDim.x) {
        unsigned char value = image[i];
        atomicAdd(&local_hist[value], 1);
    }
    // aggregate the local histograms using warp-level atomic operations in shared memory
    for (int i = lane; i < 256; i += warpSize) {
        smem_hist[i] = 0;
    }
    // iterate over bins (0-255)
    for (int i = lane; i < 256; i += warpSize) {
        unsigned int sum = 0;
        // iterate over threads in warp
        for (int j = 0; j < warpSize; j++) {
            int index = i+j*256;
//            printf("(%d)+(%d)*256 =(%d)\n", i,j,i+j*256);
            if (index < 256){
                sum += local_hist[index];
            }


//            sum += local_hist[i + j * 256]; // FIXME: here is the illegal access
        }
        atomicAdd(&smem_hist[i], sum);
    }
    __syncthreads();
    // copy the aggregated histogram to global memory
    if (lane == 0) {
        for (int i = 0; i < 256; i++) {
            atomicAdd(&histogram[i], smem_hist[i]);
        }
    }
}


void histogramCuda(unsigned char* image, long img_size, unsigned int* histogram, int hist_size) {
    int threadBlockSize = 512;

    // allocate the vectors on the GPU
    unsigned char* deviceImage = NULL;
    checkCudaCall(hipMalloc((void **) &deviceImage, img_size * sizeof(unsigned char)));
    if (deviceImage == NULL) {
        cout << "could not allocate memory!" << endl;
        return;
    }
    unsigned int* deviceHisto = NULL;
    checkCudaCall(hipMalloc((void **) &deviceHisto, hist_size * sizeof(unsigned int)));
    if (deviceHisto == NULL) {
        checkCudaCall(hipFree(deviceImage));
        cout << "could not allocate memory!" << endl;
        return;
    }

    timer kernelTime1 = timer("kernelTime1");
    timer memoryTime = timer("memoryTime");

    // copy the original vectors to the GPU
    memoryTime.start();
    checkCudaCall(hipMemcpy(deviceImage, image, img_size*sizeof(unsigned char), hipMemcpyHostToDevice));
    memoryTime.stop();

    // Paul:
    // launch differently, change threadblock-size such we dont wast threads if the image size is not a multiple of 512
    int numBlocks = (img_size + threadBlockSize - 1) / threadBlockSize;

    // execute kernel
    kernelTime1.start();
    histogramKernel<<<numBlocks, threadBlockSize>>>(deviceImage, img_size, deviceHisto, hist_size);

//    histogramKernel<<<img_size/threadBlockSize, threadBlockSize>>>(deviceImage, img_size, deviceHisto, hist_size);
    hipDeviceSynchronize();
    kernelTime1.stop();

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    memoryTime.start();
    checkCudaCall(hipMemcpy(histogram, deviceHisto, hist_size * sizeof(unsigned int), hipMemcpyDeviceToHost));
    memoryTime.stop();

    checkCudaCall(hipFree(deviceImage));
    checkCudaCall(hipFree(deviceHisto));

    cout << "histogram (kernel): \t\t" << kernelTime1  << endl;
    cout << "histogram (memory): \t\t" << memoryTime << endl;
}

void histogramSeq(unsigned char* image, long img_size, unsigned int* histogram, int hist_size) {
  int i; 

  timer sequentialTime = timer("Sequential");
  
  for (i=0; i<hist_size; i++){
      histogram[i]=0;
  }

  sequentialTime.start();
  for (i=0; i<img_size; i++) {
	histogram[image[i]]++;
  }
  sequentialTime.stop();
  
  cout << "histogram (sequential): \t\t" << sequentialTime << endl;

}

int main(int argc, char* argv[]) {
    int c;
    int seed = 42;
    const char *image_path = 0;
    image_path ="../../images/pat1_100x150.pgm";
    int gen_image = 0;
    int debug = 0;

    int num_rows = 150;
    int num_cols = 100;

    /* Read command-line options. */
    while((c = getopt(argc, argv, "s:i:rp:n:m:g")) != -1) {
        switch(c) {
            case 's':
                seed = atoi(optarg);
                break;
            case 'i':
            	image_path = optarg;
            	break;
            case 'r':
            	gen_image = 1;
            	break;
            case 'n':
            	num_rows = strtol(optarg, 0, 10);
            	break;
            case 'm':
				num_cols = strtol(optarg, 0, 10);
				break;
			case 'g':
				debug = 1;
				break;
            case '?':
                fprintf(stderr, "Unknown option character '\\x%x'.\n", optopt);
                return -1;
            default:
                return -1;
        }
    }

    int hist_size = 256;
    long img_size = num_rows*num_cols;

    unsigned char *image = (unsigned char *)malloc(img_size * sizeof(unsigned char)); 
    unsigned int *histogramS = (unsigned int *)malloc(hist_size * sizeof(unsigned int));     
    unsigned int *histogram = (unsigned int *)malloc(hist_size * sizeof(unsigned int));

    /* Seed such that we can always reproduce the same random vector */
    if (gen_image){
    	srand(seed);
    	generate_image(num_rows, num_cols, image);
    }else{
    	read_image(image_path,num_rows, num_cols, image);
    }

    histogramSeq(image, img_size, histogramS, hist_size);
    histogramCuda(image, img_size, histogram, hist_size);
    
    // verify the resuls
    for(int i=0; i<hist_size; i++) {
	  if (histogram[i]!=histogramS[i]) {
            cout << "error in results! Bin " << i << " is "<< histogram[i] << ", but should be " << histogramS[i] << endl; 
            exit(1);
        }
    }
    cout << "results OK!" << endl;
     
    free(image);
    free(histogram);
    free(histogramS);         
    
    return 0;
}
