#include "hip/hip_runtime.h"
//
// Created by Paul Hosek on 18.03.23.
//

#include "reduction_options.cuh"

// v1. simple reduction
if (threadIdx.x < 256) {
    atomicAdd(&histogram[threadIdx.x], local_hist[threadIdx.x]);
}


// v2. warp reduction
// each first warp in a thread block aggregates the values of the local hist into the global hist
// 256/32 = 8 -> 8 bins for each thread to add to the global array
if (threadIdx.x < warpSize) {
    for (int i = 0; i < 256; i += warpSize) {
        atomicAdd(&histogram[i + threadIdx.x], local_hist[i + threadIdx.x]);
    }
}

// v3. tree reduction // todo this cannot be used with shared memory, bc shared memory can only be accessed by 1 thread block
// reduce local hists into
// Perform tree-reduction on local histograms
for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (threadIdx.x < s) {
        int i = threadIdx.x * 2;
        local_hist[i] += local_hist[i + s];
    }
    __syncthreads();
}
if (threadIDX.x < 256){ // copy first local_hist, no need for atomic here
    histogram[g_tid] = local_hist[g_tid]);
}

//if (threadIdx.x == 0) {
//    for (int i = 0; i < 256; i++) {
//        atomicAdd(&histogram[i], local_hist[i]);
//    }






__global__ void histogramKernel(unsigned char* image, long img_size, unsigned int* histogram, int hist_size) {
    // Compute the global thread ID
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;

    // The first 256 threads within a block initialise the local array to 0
    __shared__ unsigned int local_hist[256];
    if (threadIdx.x < 256) {
        local_hist[threadIdx.x] = 0;
    }
    __syncthreads();


    uchar4 in;
    int stride = blockDim.x * gridDim.x;
    for (int i = tid; i < img_size/4; i +=stride) {
        in = *reinterpret_cast<uchar4*>(image[i]);
    }

    __syncthreads();
}
