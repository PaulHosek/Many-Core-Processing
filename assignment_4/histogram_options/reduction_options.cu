#include "hip/hip_runtime.h"
//
// Created by Paul Hosek on 18.03.23.
//

#include "reduction_options.cuh"

// v1. simple reduction
if (threadIdx.x < 256) {
    atomicAdd(&histogram[threadIdx.x], local_hist[threadIdx.x]);
}


// v2. warp reduction
// each first warp in a thread block aggregates the values of the local hist into the global hist
// 256/32 = 8 -> 8 bins for each thread to add to the global array
if (threadIdx.x < warpSize) {
    for (int i = 0; i < 256; i += warpSize) {
        atomicAdd(&histogram[i + threadIdx.x], local_hist[i + threadIdx.x]);
    }
}

// v3. tree reduction
// reduce local hists into
// Perform tree-reduction on local histograms
for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (threadIdx.x < s) {
        int i = threadIdx.x * 2;
        local_hist[i] += local_hist[i + s];
    }
    __syncthreads();
}
if (threadIDX.x < 256){ // copy first local_hist, no need for atomic here
    histogram[g_tid] = local_hist[g_tid]);
}

//if (threadIdx.x == 0) {
//    for (int i = 0; i < 256; i++) {
//        atomicAdd(&histogram[i], local_hist[i]);
//    }
